#include "hip/hip_runtime.h"
#include "mublas.cuh"

static __global__ void k_compute_batched_ptrs(
        const half * src0_as_f16, const half * src1_as_f16, char * dst,
        const void ** ptrs_src, void ** ptrs_dst,
        int64_t ne12, int64_t ne13,
        int64_t ne23,
        size_t  nb02, size_t  nb03,
        size_t  nb12, size_t  nb13,
        size_t  nbd2, size_t  nbd3,
        int64_t r2,   int64_t r3) {
    const int64_t i13 = blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t i12 = blockIdx.y * blockDim.y + threadIdx.y;

    if (i13 >= ne13 || i12 >= ne12) {
        return;
    }

    const int64_t i03 = i13 / r3;
    const int64_t i02 = i12 / r2;

    ptrs_src[0*ne23 + i12 + i13*ne12] = (const char *) src0_as_f16 + i02*nb02 + i03*nb03;
    ptrs_src[1*ne23 + i12 + i13*ne12] = (const char *) src1_as_f16 + i12*nb12 + i13*nb13;
    ptrs_dst[0*ne23 + i12 + i13*ne12] = (      char *)         dst + i12*nbd2 + i13*nbd3;
}

void ggml_cuda_mul_mat_batched_cublas_gemm_batched_ex(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    const half * src0_f16, const half * src1_f16, char * dst_t,
    const size_t nbd2, const size_t nbd3,
    const int64_t r2, const int64_t r3,
    const int64_t s11, const int64_t s12, const int64_t s13,
    const void * alpha, const void * beta,
    const hipDataType cu_data_type,
    const hipblasComputeType_t cu_compute_type,
    hipStream_t main_stream
) {
    GGML_TENSOR_BINARY_OP_LOCALS

    // use hipblasGemmBatchedEx
    const int64_t ne23 = ne12*ne13;

    // Allocate memory for pointer arrays using hipMalloc to avoid segmentation faults in muBLAS.
    const void ** ptrs_src;
    void ** ptrs_dst;
    CUDA_CHECK(hipMalloc((void **)&ptrs_src, sizeof(void *)*2*ne23));
    CUDA_CHECK(hipMalloc((void **)&ptrs_dst, sizeof(void *)*1*ne23));

    dim3 block_dims(ne13, ne12);
    k_compute_batched_ptrs<<<1, block_dims, 0, main_stream>>>(
            src0_f16, src1_f16, dst_t,
            ptrs_src, ptrs_dst,
            ne12, ne13,
            ne23,
            nb02, nb03,
            src1->type == GGML_TYPE_F16 ? nb12 : s12*sizeof(half),
            src1->type == GGML_TYPE_F16 ? nb13 : s13*sizeof(half),
            nbd2, nbd3,
            r2, r3);
    CUDA_CHECK(hipGetLastError());

    // This operation is essential for musa; without it, generated tokens will
    // be garbled and may eventually cause MUBLAS_STATUS_INTERNAL_ERROR.
    CUDA_CHECK(hipDeviceSynchronize());

    CUBLAS_CHECK(
    hipblasGemmBatchedEx(ctx.cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N,
            ne01, ne11, ne10,
            alpha, (const void **) (ptrs_src + 0*ne23), HIP_R_16F,   nb01/nb00,
                   (const void **) (ptrs_src + 1*ne23), HIP_R_16F,   s11,
            beta,  (      void **) (ptrs_dst + 0*ne23), cu_data_type, ne0,
            ne23,
            cu_compute_type,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    CUDA_CHECK(hipFree(ptrs_src));
    CUDA_CHECK(hipFree(ptrs_dst));
}
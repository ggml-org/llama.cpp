#include "hip/hip_runtime.h"
#include "softcap.cuh"

static __global__ void softcap_f32(const float * x, float * dst, const float scale, const float softcap, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = tanhf(scale * x[i]) * softcap;
}

static void softcap_f32_cuda(const float * x, float * dst, const float scale, const float softcap, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SOFTCAP_BLOCK_SIZE - 1) / CUDA_SOFTCAP_BLOCK_SIZE;
    softcap_f32<<<num_blocks, CUDA_SOFTCAP_BLOCK_SIZE, 0, stream>>>(x, dst, scale, softcap, k);
}

void ggml_cuda_op_softcap(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float scale;
    float softcap;
    memcpy(&softcap, (float *) dst->op_params + 0, sizeof(float));
    memcpy(&scale,   (float *) dst->op_params + 1, sizeof(float));

    softcap_f32_cuda(src0_d, dst_d, scale, softcap, ggml_nelements(src0), stream);
}

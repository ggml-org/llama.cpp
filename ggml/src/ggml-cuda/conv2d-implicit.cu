#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
#include "ggml.h"
#include "common.cuh"
#include "convert.cuh"
#include "conv2d-implicit.cuh"


typedef unsigned int uint;
constexpr uint WARPSIZE = 32;
#define CUDA_NCHW_2_NHWC_TILE_DIM 32
#define CUDA_NCHW_2_NHWC_BLOCK_NM 8
#define CUDA_NCHW_2_NHWC_BLOCK_ROWS 8


//currently not use; in future for split-k kernels
// static __global__ void reduce_f32(const float * __restrict__ x, float * __restrict__ dst, const int ncols, const int nrows) {
//     const int row = blockIdx.x;
//     const int col = threadIdx.x;

//     float     sum        = 0.0f;
//     if (row * blockDim.x + col < ncols) {
//         for (int i = 0; i < nrows; ++i){
//             sum += x[i * ncols + row * blockDim.x + col];
//         }
//         dst[row * blockDim.x + col] = sum;
//     }
// }

template <typename src_T, typename dst_T>
static __global__ void NCHW2NHWC(const src_T *src, dst_T * dst, const int ne, const int ne00, const int ne01){

    const int64_t nmat = ne / (ne00 * ne01);
    const int64_t n = ne00 * ne01;

    int x  = blockIdx.x * CUDA_NCHW_2_NHWC_TILE_DIM + threadIdx.x;
    int y  = blockIdx.y * CUDA_NCHW_2_NHWC_TILE_DIM + threadIdx.y;
    int tx = blockIdx.y * CUDA_NCHW_2_NHWC_TILE_DIM + threadIdx.x;  // transpose block offset
    int ty = blockIdx.x * CUDA_NCHW_2_NHWC_TILE_DIM + threadIdx.y;

    __shared__ src_T tile[CUDA_NCHW_2_NHWC_TILE_DIM][CUDA_NCHW_2_NHWC_TILE_DIM];

    for(int i = 0; i < CUDA_NCHW_2_NHWC_BLOCK_NM; ++i){

        const unsigned int imat = blockIdx.z * CUDA_NCHW_2_NHWC_BLOCK_NM + i;
        if(imat >= nmat)
            break;
        for (int j = 0; j < CUDA_NCHW_2_NHWC_TILE_DIM; j += CUDA_NCHW_2_NHWC_BLOCK_ROWS){
            if(x < ne01 && y + j < ne00){
                const int row = threadIdx.y+j;
                const int col = threadIdx.x ^ row;
                tile[row][col] = src[imat*n + (y+j)*ne01 + x];
            }
        }
        __syncthreads();

        for (int j = 0; j < CUDA_NCHW_2_NHWC_TILE_DIM; j += CUDA_NCHW_2_NHWC_BLOCK_ROWS){
            if(ty + j < ne01 && tx < ne00){
                const int col = (threadIdx.y+j) ^ threadIdx.x;
                dst[imat*n + (ty+j)*ne00 + tx] = ggml_cuda_cast<dst_T>(tile[threadIdx.x][col]);
            }
        }
    }
}

template<typename T, const int BM, const int BN, const int BK, const int WM, const int WN,
          const int WNITER, const int TM, const int TN, const int NUM_THREADS,
          // layout: 0, NHWC; 1, NCHW
          const int layout, const bool vec_load, const int ksplit, const int PAD=4>
static __global__ void conv2d_implicit_kernel(const float * __restrict__ input,
                                              const T * __restrict__ kernel,
                                              float * __restrict__ output,
                                              const param_t param) {

    __shared__ char smem[sizeof(float) * (TM*TN*NUM_THREADS) <= sizeof(float) * 2 * (BM+PAD) * BK +  sizeof(T)*2*BK * (BN+PAD) ?
                         sizeof(float)*2*(BM+PAD)*BK + sizeof(T)*2*BK*(BN+PAD) : sizeof(float) * (TM*TN*NUM_THREADS)];
    T *smemweight = reinterpret_cast<T *>(smem);
    float *smeminput = reinterpret_cast<float *>(smem + 2 * BK * (BN+PAD) * sizeof(T));

    const uint tx = threadIdx.x;
    const uint bx = blockIdx.x;
    const uint by = blockIdx.y;

    const uint PQ = param.Oh * param.Ow;

    // Warp tile
    const uint lane_id = tx % WARPSIZE;
    const uint warp_id = tx / WARPSIZE;
    const int mma_tid_x = warp_id / (BN / WN);
    const int mma_tid_y = warp_id % (BN / WN);

    // size of the warp subtile
    constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
    constexpr uint WSUBM = WM / WMITER; // 64/2=32
    constexpr uint WSUBN = WN / WNITER; // 32/2=16

    // Placement of the thread in the warp subtile
    const uint threadColInWarp = lane_id % (WSUBN / TN); // i%(16/4)
    const uint threadRowInWarp = lane_id / (WSUBN / TN); // i/4

    int z = blockIdx.z;

    int inChannelOffset = layout == 0 ? param.c * param.w : param.h * param.w;
    int weightKOffset = param.c * param.r * param.s;

    const uint ks =  (ksplit > 0) ? (weightKOffset + ksplit - 1) / ksplit : weightKOffset;
    const uint start_k = (ksplit > 0)? z * ks: 0;
    const uint end_k = min(start_k + ks, weightKOffset);

    int write_flag = 1;
    T weight_frag[2][WNITER * TN];
    float input_frag[2][WMITER * TM] = {0.f};
    float output_frag[WMITER * TM * WNITER * TN] = {0.f};

    // calculating the indices that this thread will load into SMEM
    // we'll load 128bit / 32bit = 4 elements per thread at each step
    const uint innerRowA = tx / (BK / 4);
    const uint innerColA = tx % (BK / 4);
    constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;

// ldg
    const uint weight_sts_addr = innerRowA + innerColA * (BN+PAD) * 4;
#pragma unroll
    for (uint offset = 0; offset + rowStrideA <= BN; offset += rowStrideA) {
        if(vec_load){
            if (by * BN  + innerRowA + offset < param.k &&   start_k + innerColA * 4 < end_k){
                if constexpr (std::is_same_v<T, float>){
                    float4 tmp = reinterpret_cast<const float4 *>(&kernel[(by * BN + innerRowA + offset) * weightKOffset + start_k + innerColA * 4])[0];
                    smemweight[weight_sts_addr + offset +          0] = tmp.x;
                    smemweight[weight_sts_addr + offset +   (BN+PAD)] = tmp.y;
                    smemweight[weight_sts_addr + offset + 2*(BN+PAD)] = tmp.z;
                    smemweight[weight_sts_addr + offset + 3*(BN+PAD)] = tmp.w;
                }else{ // read 4 halves
                    float2 tmp = reinterpret_cast<const float2 *>(&kernel[(by * BN + innerRowA + offset) * weightKOffset + start_k + innerColA * 4])[0];
                    const half *val = reinterpret_cast<const half *>(&tmp);
                    smemweight[weight_sts_addr + offset +          0] = val[0];
                    smemweight[weight_sts_addr + offset +   (BN+PAD)] = val[1];
                    smemweight[weight_sts_addr + offset + 2*(BN+PAD)] = val[2];
                    smemweight[weight_sts_addr + offset + 3*(BN+PAD)] = val[3];
                }
            } else {
#pragma unroll
                for (int i = 0; i < 4; ++i){
                    smemweight[weight_sts_addr + offset + i*(BN+PAD)] = (T)0.f;
                }
            }
        }else{
#pragma unroll
            for (int i = 0; i < 4; ++i){
                if (by * BN  + innerRowA + offset < param.k &&  start_k + innerColA * 4 + i < end_k){
                    smemweight[weight_sts_addr + offset + i*(BN+PAD)] = kernel[(by * BN + innerRowA + offset) * weightKOffset + start_k + innerColA * 4 + i];
                } else {
                    smemweight[weight_sts_addr + offset + i*(BN+PAD)] = (T)0.f;
                }
            }
        }
    }


    const uint input_sts_addr = innerRowA + innerColA * (BM+PAD) * 4;
#pragma unroll
    for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
        int n = (ksplit > 0) ? (bx * BM + innerRowA + offset) / PQ : z;
        const unsigned int npq_res = (bx * BM + innerRowA + offset) % PQ;
        const int posh_ori = fastdiv((ksplit > 0) ? npq_res: bx * BM + innerRowA + offset, param.OW_fastdiv) * param.u - param.p;
        const int posw_ori = fastmodulo((ksplit > 0) ? npq_res: bx * BM + innerRowA + offset, param.OW_fastdiv) * param.v - param.q;
        int inOffset = n * param.c * param.h * param.w ;
        if(vec_load){
            const uint cur0 = fastdiv(start_k + innerColA * 4,
                   layout == 0 ? param.SC_fastdiv : param.RS_fastdiv);             // channel offset
            const uint cur1 = fastdiv(fastmodulo(start_k + innerColA * 4,
                layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
            const uint cur2 = fastmodulo(fastmodulo(start_k + innerColA * 4,
                layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
            const uint curC = layout == 0 ? cur2 : cur0;
            const uint curR = layout == 0 ? cur0 : cur1;
            const uint curS = layout == 0 ? cur1 : cur2;
            const int curH = posh_ori + curR * param.d_h; // input h
            const int curW = posw_ori + curS * param.d_w; // input w
            if (curH >= 0 && curW >= 0 && curW < param.w && curH < param.h && start_k + innerColA * 4 < end_k){
                int inOffsetTmp = layout == 0 ?
                                curH * inChannelOffset + curW * param.c + curC:
                                curC * inChannelOffset + curH * param.w + curW;
                float4 tmp = reinterpret_cast<const float4 *>(&input[inOffset + inOffsetTmp])[0];
                smeminput[input_sts_addr + offset +           0] = tmp.x;
                smeminput[input_sts_addr + offset +      BM+PAD] = tmp.y;
                smeminput[input_sts_addr + offset +  2*(BM+PAD)] = tmp.z;
                smeminput[input_sts_addr + offset +  3*(BM+PAD)] = tmp.w;
            } else {
#pragma unroll
                for (int i = 0; i < 4; ++i)
                    smeminput[input_sts_addr + offset + i*(BM+PAD)] = 0.f;
            }
        } else {
#pragma unroll
            for (int i = 0; i < 4; ++i){
                const uint cur0 = fastdiv(start_k + innerColA * 4 + i,
                    layout == 0 ? param.SC_fastdiv : param.RS_fastdiv);             // channel offset
                const uint cur1 = fastdiv(fastmodulo(start_k + innerColA * 4 + i,
                    layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                    layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
                const uint cur2 = fastmodulo(fastmodulo(start_k + innerColA * 4 + i,
                    layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                    layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
                const uint curC = layout == 0 ? cur2 : cur0;
                const uint curR = layout == 0 ? cur0 : cur1;
                const uint curS = layout == 0 ? cur1 : cur2;
                const int curH = posh_ori + curR * param.d_h; // input h
                const int curW = posw_ori + curS * param.d_w; // input w
                if (curH >= 0 && curW >= 0 && curW < param.w && curH < param.h && start_k + innerColA * 4 + i < end_k){
                    int inOffsetTmp = layout == 0 ?
                                curH * inChannelOffset + curW * param.c + curC:
                                curC * inChannelOffset + curH * param.w + curW;
                    smeminput[input_sts_addr + offset + i*(BM+PAD)] = input[inOffset + inOffsetTmp];
                } else {
                    smeminput[input_sts_addr + offset + i*(BM+PAD)] = 0.f;
                }
            }
        }
    }
    __syncthreads();

    // lds
    const uint input_lds_addr =  mma_tid_x * WM;
#pragma unroll
    for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx)
#pragma unroll
      for (uint i = 0; i < TM; ++i)
        input_frag[0][wSubRowIdx * TM + i] = smeminput[input_lds_addr + wSubRowIdx * WSUBM +
                               threadRowInWarp * TM + i];

    const uint weight_lds_addr = mma_tid_y * WN;
#pragma unroll
    for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx)
#pragma unroll
      for (uint i = 0; i < TN; ++i)
        weight_frag[0][wSubColIdx * TN + i] = smemweight[weight_lds_addr + wSubColIdx * WSUBN +
                             threadColInWarp * TN + i];

    for (int crs = start_k; crs < end_k; crs += BK) {

        int load_flag = write_flag ^ 1;
#pragma unroll
        for (int subcrs = 0; subcrs < BK - 1; ++subcrs)
        {

#pragma unroll
            for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx)
#pragma unroll
                for (uint i = 0; i < TN; ++i)
                    weight_frag[(subcrs + 1) % 2][wSubColIdx * TN + i] = smemweight[load_flag * (BN+PAD) * BK +
                        (subcrs + 1) * (BN+PAD) + weight_lds_addr + wSubColIdx * WSUBN + threadColInWarp * TN + i];
#pragma unroll
            for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx)
#pragma unroll
                for (uint i = 0; i < TM; ++i)
                    input_frag[(subcrs + 1) % 2][wSubRowIdx * TM + i] = smeminput[load_flag * (BM+PAD) * BK +
                        (subcrs + 1) * (BM+PAD) + input_lds_addr + wSubRowIdx * WSUBM + threadRowInWarp * TM + i];

            // execute warptile matmul
#pragma unroll
            for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
#pragma unroll
                for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
                    // calculate per-thread results
#pragma unroll
                    for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
#pragma unroll
                        for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                            output_frag[(wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
                                        (wSubColIdx * TN) + resIdxN] +=
                                input_frag[subcrs % 2][wSubRowIdx * TM + resIdxM] *
                                ggml_cuda_cast<float>(weight_frag[subcrs % 2][wSubColIdx * TN + resIdxN]);
                        }
                    }
                }
            }
        }
        // ldg
#pragma unroll
        for (uint offset = 0; offset + rowStrideA <= BN; offset += rowStrideA) {
            if(vec_load){
                if (by * BN  + innerRowA + offset < param.k &&  innerColA * 4 + crs + BK < end_k){
                    if constexpr (std::is_same_v<T, float>){
                        float4 tmp = reinterpret_cast<const float4 *>(&kernel[(by * BN + innerRowA + offset) * weightKOffset + innerColA * 4 + crs + BK])[0];
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset +          0] = tmp.x;
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset +   (BN+PAD)] = tmp.y;
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + 2*(BN+PAD)] = tmp.z;
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + 3*(BN+PAD)] = tmp.w;
                    } else {
                        float2 tmp = reinterpret_cast<const float2 *>(&kernel[(by * BN + innerRowA + offset) * weightKOffset + innerColA * 4 + crs + BK])[0];
                        const half *val = reinterpret_cast<const half *>(&tmp);
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset +          0] = val[0];
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset +   (BN+PAD)] = val[1];
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + 2*(BN+PAD)] = val[2];
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + 3*(BN+PAD)] = val[3];
                    }
                } else {
#pragma unroll
                    for (int i = 0; i < 4; ++i)
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + i*(BN+PAD)] = (T)0.f;
                }
            }else{
#pragma unroll
                for (int i = 0; i < 4; ++i){
                    if (by * BN  + innerRowA + offset < param.k &&  innerColA * 4 + crs + BK + i < end_k){
                        // float4 tmp = reinterpret_cast<float4 *>(&param.weight[(by * BN + innerRowA + offset) * weightKOffset + innerColA * 4 + crs + BK + i])[0];
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + i*(BN+PAD)] = kernel[(by * BN + innerRowA + offset) * weightKOffset + innerColA * 4 + crs + BK + i];
                    } else {
                        smemweight[write_flag * (BN+PAD) * BK + weight_sts_addr + offset + i*(BN+PAD)] = (T)0.f;
                    }
                }
            }
        }
#pragma unroll
        for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
            int n = (ksplit > 0) ? (bx * BM + innerRowA + offset) / PQ : z;
            const unsigned int npq_res = (bx * BM + innerRowA + offset) % PQ;
            const int posh_ori = fastdiv((ksplit > 0) ? npq_res: bx * BM + innerRowA + offset, param.OW_fastdiv) * param.u - param.p;
            const int posw_ori = fastmodulo((ksplit > 0) ? npq_res: bx * BM + innerRowA + offset, param.OW_fastdiv) * param.v - param.q;
            int inOffset = n * param.c * param.h * param.w ;
            if(vec_load){
                const uint cur0 = fastdiv(innerColA * 4 + crs + BK,
                    layout == 0 ? param.SC_fastdiv : param.RS_fastdiv);             // channel offset
                const uint cur1 = fastdiv(fastmodulo(innerColA * 4 + crs + BK,
                    layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                    layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
                const uint cur2 = fastmodulo(fastmodulo(innerColA * 4 + crs + BK,
                    layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                    layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
                const uint curC = layout == 0 ? cur2 : cur0;
                const uint curR = layout == 0 ? cur0 : cur1;
                const uint curS = layout == 0 ? cur1 : cur2;

                const int curH = posh_ori + curR * param.d_h; // input h
                const int curW = posw_ori + curS * param.d_w; // input w
                if (curH >= 0 && curW >= 0 && curW < param.w && curH < param.h && innerColA * 4 + crs + BK < end_k){
                    // int inOffsetTmp = curH * inChannelOffset + curW * param.c + curC;
                    int inOffsetTmp = layout == 0 ?
                                curH * inChannelOffset + curW * param.c + curC:
                                curC * inChannelOffset + curH * param.w + curW;
                    float4 tmp = reinterpret_cast<const float4 *>(&input[inOffset + inOffsetTmp])[0];
                    smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset +           0] = tmp.x;
                    smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset +      BM+PAD] = tmp.y;
                    smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset +  2*(BM+PAD)] = tmp.z;
                    smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset +  3*(BM+PAD)] = tmp.w;
                } else {
#pragma unroll
                    for (int i = 0; i < 4; ++i)
                        smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset + i*(BM+PAD)] = 0.f;
                }
            } else {
#pragma unroll
                for (int i = 0; i < 4; ++i){
                    const uint cur0 = fastdiv(innerColA * 4 + crs + BK + i,
                        layout == 0 ? param.SC_fastdiv : param.RS_fastdiv);             // channel offset
                    const uint cur1 = fastdiv(fastmodulo(innerColA * 4 + crs + BK + i,
                        layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                        layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
                    const uint cur2 = fastmodulo(fastmodulo(innerColA * 4 + crs + BK + i,
                        layout == 0 ? param.SC_fastdiv : param.RS_fastdiv),
                        layout == 0 ? param.C_fastdiv  : param.S_fastdiv); // kernel r offset
                    const uint curC = layout == 0 ? cur2 : cur0;
                    const uint curR = layout == 0 ? cur0 : cur1;
                    const uint curS = layout == 0 ? cur1 : cur2;

                    const int curH = posh_ori + curR * param.d_h; // input h
                    const int curW = posw_ori + curS * param.d_w; // input w
                    if (curH >= 0 && curW >= 0 && curW < param.w && curH < param.h && innerColA * 4 + crs + BK + i < end_k){
                        int inOffsetTmp = layout == 0 ?
                                curH * inChannelOffset + curW * param.c + curC:
                                curC * inChannelOffset + curH * param.w + curW;
                        smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset + i*(BM+PAD)] = input[inOffset + inOffsetTmp];
                    } else {
                        smeminput[write_flag * (BM+PAD) * BK + input_sts_addr + offset + i*(BM+PAD)] = 0.f;
                    }
                }
            }
        }
        __syncthreads();

        write_flag ^= 1;

#pragma unroll
        for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx)
#pragma unroll
            for (uint i = 0; i < TM; ++i)
                input_frag[0][wSubRowIdx * TM + i] = smeminput[(load_flag ^ 1) * (BM+PAD) * BK +
                    input_lds_addr + wSubRowIdx * WSUBM + threadRowInWarp * TM + i];
#pragma unroll
        for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx)
#pragma unroll
            for (uint i = 0; i < TN; ++i)
                weight_frag[0][wSubColIdx * TN + i] = smemweight[(load_flag ^ 1) * (BN+PAD) * BK +
                    weight_lds_addr + wSubColIdx * WSUBN + threadColInWarp * TN + i];
#pragma unroll
        for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
#pragma unroll
            for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
                // calculate per-thread results
#pragma unroll
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
#pragma unroll
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                        output_frag[(wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
                                    (wSubColIdx * TN) + resIdxN] +=
                            input_frag[1][wSubRowIdx * TM + resIdxM] *
                            ggml_cuda_cast<float>(weight_frag[1][wSubColIdx * TN + resIdxN]);
                    }
                }
            }
        }
    }

    // reuse smem
    float *smemoutput = reinterpret_cast<float *>(smem);

    const uint output_lds_addr = warp_id * WSUBM * WSUBN + lane_id;
    const uint output_sts_addr = mma_tid_x * BN / WN * TM * TN * WARPSIZE + mma_tid_y * TM * TN * WARPSIZE +
                         threadColInWarp * TN * WSUBM + threadRowInWarp * TM;
    const uint m_idx = by * BN + mma_tid_y * WN;
    const uint n_idx = bx * BM + mma_tid_x * WM;

#pragma unroll
    for (int i = 0; i < WMITER; ++i)
    {
#pragma unroll
        for (int j = 0; j < WNITER; ++j)
        {
            __syncthreads();

#pragma unroll
            for (int subi = 0; subi < TM; ++subi)
            {
#pragma unroll
                for (int subj = 0; subj < TN; ++subj)
                {
                    // output sts
                    smemoutput[output_sts_addr + subj * WSUBM + subi] =
                        output_frag[(i * TM + subi) * (WNITER * TN) + j * TN + subj];
                }
            }
            __syncthreads();
#pragma unroll
            for (int subk = 0; subk < TM * TN; ++subk){
                const uint row =  m_idx + j * WSUBN + (lane_id + subk * WARPSIZE) / WSUBM;
                const uint gemm_i =  n_idx + i * WSUBM + (lane_id + subk * WARPSIZE) % WSUBM;
                const int n = (ksplit > 0) ? gemm_i / PQ : z;
                const int col = (ksplit > 0) ? gemm_i % PQ : gemm_i;
                if (n < param.n && row < param.k && col < param.Oh * param.Ow){
                    const uint outOffset = ksplit > 0 ?
                                z * param.n * param.k * param.Oh * param.Ow + n * param.k * param.Oh * param.Ow +
                                row * param.Oh * param.Ow + col :
                                z * param.k * param.Oh * param.Ow + row * param.Oh * param.Ow + col;
                    output[outOffset] = smemoutput[output_lds_addr + subk * WARPSIZE];
                }
            }
        }
    }
}



template <unsigned int mma_tiles_per_warp_m, unsigned int mma_tiles_per_warp_k, unsigned int smem_stride>
__device__ __forceinline__ void ldmatrix_a(
  const half* src,
  half (&reg)[mma_tiles_per_warp_m][mma_tiles_per_warp_k][4]
){
#if __CUDA_ARCH__ >= GGML_CUDA_CC_TURING
  static_assert(mma_tiles_per_warp_m == 8, "mma_tiles_per_warp_m must be 4");
  static_assert(mma_tiles_per_warp_k == 4, "mma_tiles_per_warp_k must be 4");

  uint32_t (&reg_) [mma_tiles_per_warp_m][mma_tiles_per_warp_k][2] = reinterpret_cast<uint32_t(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_k][2]>(reg);
  unsigned int logical_offset = (threadIdx.x % 32) * smem_stride;
  unsigned int swizzled_offset = logical_offset ^ ((logical_offset & 0b10000000) >> 4);
  swizzled_offset = swizzled_offset ^ ((swizzled_offset & 0b1100000) >> 2);
  uint32_t src_addr = cvta_to_shared_u32(src + swizzled_offset);
  constexpr unsigned int smem_stride_ = smem_stride * sizeof(half); // convert stride to bytes

    // 0
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[0][0][0]), "=r"(reg_[0][0][1]), "=r"(reg_[1][0][0]), "=r"(reg_[1][0][1])
      : "r"(src_addr)
    );

    // 0
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[2][0][0]), "=r"(reg_[2][0][1]), "=r"(reg_[3][0][0]), "=r"(reg_[3][0][1])
      : "r"(src_addr + 32 * smem_stride_)
    );

    // 0
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[4][0][0]), "=r"(reg_[4][0][1]), "=r"(reg_[5][0][0]), "=r"(reg_[5][0][1])
      : "r"(src_addr + 64 * smem_stride_)
    );

    // 0
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[6][0][0]), "=r"(reg_[6][0][1]), "=r"(reg_[7][0][0]), "=r"(reg_[7][0][1])
      : "r"(src_addr + 96 * smem_stride_)
    );

    src_addr ^= 0b10000;

    // 1
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[0][1][0]), "=r"(reg_[0][1][1]), "=r"(reg_[1][1][0]), "=r"(reg_[1][1][1])
      : "r"(src_addr)
    );

    // 1
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[2][1][0]), "=r"(reg_[2][1][1]), "=r"(reg_[3][1][0]), "=r"(reg_[3][1][1])
      : "r"(src_addr + 32 * smem_stride_)
    );

    // 1
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[4][1][0]), "=r"(reg_[4][1][1]), "=r"(reg_[5][1][0]), "=r"(reg_[5][1][1])
      : "r"(src_addr + 64 * smem_stride_)
    );

    // 1
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[6][1][0]), "=r"(reg_[6][1][1]), "=r"(reg_[7][1][0]), "=r"(reg_[7][1][1])
      : "r"(src_addr + 96 * smem_stride_)
    );

    src_addr ^= 0b110000;

    // 2
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[0][2][0]), "=r"(reg_[0][2][1]), "=r"(reg_[1][2][0]), "=r"(reg_[1][2][1])
      : "r"(src_addr)
    );

    // 2
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[2][2][0]), "=r"(reg_[2][2][1]), "=r"(reg_[3][2][0]), "=r"(reg_[3][2][1])
      : "r"(src_addr + 32 * smem_stride_)
    );

    // 2
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[4][2][0]), "=r"(reg_[4][2][1]), "=r"(reg_[5][2][0]), "=r"(reg_[5][2][1])
      : "r"(src_addr + 64 * smem_stride_)
    );

    // 2
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[6][2][0]), "=r"(reg_[6][2][1]), "=r"(reg_[7][2][0]), "=r"(reg_[7][2][1])
      : "r"(src_addr + 96 * smem_stride_)
    );
    src_addr ^= 0b10000;

    // 3
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[0][3][0]), "=r"(reg_[0][3][1]), "=r"(reg_[1][3][0]), "=r"(reg_[1][3][1])
      : "r"(src_addr)
    );

    // 3
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[2][3][0]), "=r"(reg_[2][3][1]), "=r"(reg_[3][3][0]), "=r"(reg_[3][3][1])
      : "r"(src_addr + 32 * smem_stride_)
    );

    // 3
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[4][3][0]), "=r"(reg_[4][3][1]), "=r"(reg_[5][3][0]), "=r"(reg_[5][3][1])
      : "r"(src_addr + 64 * smem_stride_)
    );

    // 3
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[6][3][0]), "=r"(reg_[6][3][1]), "=r"(reg_[7][3][0]), "=r"(reg_[7][3][1])
      : "r"(src_addr + 96 * smem_stride_)
    );
#else
    GGML_UNUSED(src);
    GGML_UNUSED(reg);
    NO_DEVICE_CODE;
#endif
}

template <unsigned int mma_tiles_per_warp_k, unsigned int mma_tiles_per_warp_n, unsigned int smem_stride>
__device__ __forceinline__ void ldmatrix_b(
  const half* src,
  half (&reg)[mma_tiles_per_warp_k][mma_tiles_per_warp_n][2]
){
#if __CUDA_ARCH__ >= GGML_CUDA_CC_TURING

  static_assert(mma_tiles_per_warp_k == 4, "mma_tiles_per_warp_k must be 4");
  static_assert(mma_tiles_per_warp_n == 8, "mma_tiles_per_warp_n must be 8");

  uint32_t (&reg_) [4][8] = reinterpret_cast<uint32_t(&)[4][8]>(reg);
  unsigned int logical_offset = (threadIdx.x % 32) * smem_stride;
  unsigned int swizzled_offset = logical_offset ^ ((logical_offset & 0b10000000) >> 4);
  swizzled_offset = swizzled_offset ^ ((swizzled_offset & 0b1100000) >> 2);
  uint32_t src_addr = cvta_to_shared_u32(src + swizzled_offset);
  constexpr unsigned int smem_stride_ = smem_stride * sizeof(half); // convert stride to bytes

    // 0
  asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[0][0]), "=r"(reg_[0][1]), "=r"(reg_[0][2]), "=r"(reg_[0][3])
      : "r"(src_addr)
    );


  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[0][4]), "=r"(reg_[0][5]), "=r"(reg_[0][6]), "=r"(reg_[0][7])
    : "r"(src_addr + 32 * smem_stride_)
  );

  src_addr ^= 0b10000;

  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[1][0]), "=r"(reg_[1][1]), "=r"(reg_[1][2]), "=r"(reg_[1][3])
    : "r"(src_addr)
  );

  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[1][4]), "=r"(reg_[1][5]), "=r"(reg_[1][6]), "=r"(reg_[1][7])
    : "r"(src_addr + 32 * smem_stride_)
  );

  src_addr ^= 0b110000;

  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[2][0]), "=r"(reg_[2][1]), "=r"(reg_[2][2]), "=r"(reg_[2][3])
    : "r"(src_addr)
  );

  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[2][4]), "=r"(reg_[2][5]), "=r"(reg_[2][6]), "=r"(reg_[2][7])
    : "r"(src_addr + 32 * smem_stride_)
  );

  src_addr ^= 0b10000;

  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[3][0]), "=r"(reg_[3][1]), "=r"(reg_[3][2]), "=r"(reg_[3][3])
    : "r"(src_addr)
  );

  asm volatile (
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
    "{%0, %1, %2, %3}, [%4];"
    : "=r"(reg_[3][4]), "=r"(reg_[3][5]), "=r"(reg_[3][6]), "=r"(reg_[3][7])
    : "r"(src_addr + 32 * smem_stride_)
  );
#else
    GGML_UNUSED(src);
    GGML_UNUSED(reg);
    NO_DEVICE_CODE;
#endif
}

template<const int BM, const int BN, const int BK, const int WM, const int WN,
        const int WK,  const int NUM_THREADS>
static __global__ void conv2d_implicit_kernel(const half * __restrict__ input,
                                              const half * __restrict__ kernel,
                                              half * __restrict__ output,
                                              const param_t param) {
#if __CUDA_ARCH__ >= GGML_CUDA_CC_TURING

constexpr unsigned int MMA_M = 16;
constexpr unsigned int MMA_N = 8;


  const unsigned int K = param.c * param.r * param.s;
  const uint inChannelOffset = param.c * param.w;
  const uint weightKOffset = param.c * param.r * param.s;

  // loop bounds, constexpr where possible allows for loop unrolling
  constexpr unsigned int mma_tiles_per_warp_k = 4;
  constexpr unsigned int mma_tiles_per_warp_m = WM / MMA_M;
  constexpr unsigned int mma_tiles_per_warp_n = WN / MMA_N;
  const unsigned int num_block_tiles_k = (K + (BK-1)) / BK;

  // calculate block/warp indices
  const unsigned int block_m = blockIdx.y;
  const unsigned int block_n = blockIdx.x;
  const unsigned int warp_m = threadIdx.y;
  const unsigned int warp_n = threadIdx.x / 32;

  // double buffering
  extern __shared__ half shmem[];
  half* A_block_smem = shmem;
  half* B_block_smem = &shmem[BM * BK];
  constexpr int BUFFER_SIZE = BM * BK + BK * BN;

  // declare register storage
  // ptx instructions expect uint32_t registers, where each uint32_t is 2 halfs packed together
  uint32_t acc_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][2];
  uint32_t A_register[mma_tiles_per_warp_m][mma_tiles_per_warp_k][2];
  uint32_t B_register[mma_tiles_per_warp_k][mma_tiles_per_warp_n];

  // convenience cast to half for register storage
  half (&acc_register_) [mma_tiles_per_warp_m][mma_tiles_per_warp_n][4] = reinterpret_cast<half(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4]>(acc_register);
  half (&A_register_) [mma_tiles_per_warp_m][mma_tiles_per_warp_k][4] = reinterpret_cast<half(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_k][4]>(A_register);
  half (&B_register_) [mma_tiles_per_warp_k][mma_tiles_per_warp_n][2] = reinterpret_cast<half(&)[mma_tiles_per_warp_k][mma_tiles_per_warp_n][2]>(B_register);

  // accumulators start at 0
  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++){
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++){
        acc_register_[mma_m][mma_n][0] = 0;
        acc_register_[mma_m][mma_n][1] = 0;
        acc_register_[mma_m][mma_n][2] = 0;
        acc_register_[mma_m][mma_n][3] = 0;
      }
  }

  static_assert(BM == 256);
  static_assert(BN == 256);
  static_assert(BK == 32);
  static_assert(NUM_THREADS == 256);
  float4 A_gmem_cache_reg[4];
  float4 B_gmem_cache_reg[4];

  // prefetch the first block tile of A,B into shared memory

  const half* A_block_gmem = input;
  const half* B_block_gmem = kernel + block_n * BN * weightKOffset;
  tileMemcpySwizzleA<BM, NUM_THREADS>(A_block_gmem, A_block_smem, inChannelOffset, param);
  tileMemcpySwizzleB<BN, NUM_THREADS>(B_block_gmem, B_block_smem, weightKOffset, param);

  int offset_direction = 1;

  for (unsigned int block_k = 1; block_k <= num_block_tiles_k; block_k++){
    __syncthreads();

    if (block_k != num_block_tiles_k){
      const half* A_block_gmem = input;
      const half* B_block_gmem = kernel + (block_n * BN * weightKOffset);
      tileMemcpyLoadA<BM, BK, NUM_THREADS, 4>(A_block_gmem, A_gmem_cache_reg, block_k * BK, inChannelOffset, param);
      tileMemcpyLoadB<BN, BK, NUM_THREADS, 4>(B_block_gmem, B_gmem_cache_reg, block_k * BK, weightKOffset, param);
    }
    half* A_warp_tile = A_block_smem + (warp_m * WM * BK);
    half* B_warp_tile = B_block_smem + (warp_n * WN * BK);

    ldmatrix_a<mma_tiles_per_warp_m, mma_tiles_per_warp_k, BK>(A_warp_tile, A_register_);
    ldmatrix_b<mma_tiles_per_warp_k, mma_tiles_per_warp_n, BK>(B_warp_tile, B_register_);

    // outer product between mma tiles
#pragma unroll
    for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++){
#pragma unroll
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++){
#pragma unroll
        for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++){
          asm volatile (
            "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 "
            "{%0, %1}, "
            "{%2, %3}, "
            "{%4}, "
            "{%5, %6};"
            : "=r"(acc_register[mma_m][mma_n][0]), "=r"(acc_register[mma_m][mma_n][1])
            : "r"(A_register[mma_m][mma_k][0]), "r"(A_register[mma_m][mma_k][1]),
              "r"(B_register[mma_k][mma_n])
              "r"(acc_register[mma_m][mma_n][0]), "r"(acc_register[mma_m][mma_n][1])
          );
        }
      }
    }


    if (block_k != num_block_tiles_k)
    {
      // switch smem buffers each iteration
      A_block_smem = A_block_smem + BUFFER_SIZE * offset_direction;
      B_block_smem = B_block_smem + BUFFER_SIZE * offset_direction;
      offset_direction = -1 * offset_direction;

      tileMemcpySwizzleStore<BM, NUM_THREADS, 4>(A_gmem_cache_reg, A_block_smem);
      tileMemcpySwizzleStore<BN, NUM_THREADS, 4>(B_gmem_cache_reg, B_block_smem);
    }
  }

    // reuse smem
    half *smemoutput = shmem;
    const uint lane_id = threadIdx.x % WARPSIZE;
    const uint mma_row = lane_id / 4;
    const uint mma_col = lane_id % 4;
    const uint output_lds_addr = warp_m * WM * BN/2 + lane_id * BN/2 + warp_n * WN/2;
    const uint output_sts_addr = warp_m * WM * BN/2 + mma_row * BN/2 + warp_n * WN/2  + mma_col * 2;
    const uint m_idx = block_n * BN + warp_n * WN;
    const uint n_idx = block_m * BM + warp_m * WM + lane_id;

#pragma unroll
    for (int i = 0; i < 2; ++i)
    {
        __syncthreads();

        for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
        {
            for (unsigned int mma_n = i * mma_tiles_per_warp_n/2; mma_n < (i+1)*mma_tiles_per_warp_n/2; mma_n++)
            {
                uint32_t (&reg_)[2] = reinterpret_cast<uint32_t(&)[2]>(acc_register_[mma_m][mma_n]);
                uint idx = output_sts_addr +
                            mma_m * MMA_M * BN / 2 + (mma_n - i * mma_tiles_per_warp_n/2) * MMA_N;
                idx = idx ^ ((idx & 0b1110000000) >> 4);
                uint32_t* dst_ptr = reinterpret_cast<uint32_t*>(&smemoutput[idx]);
                dst_ptr[0] = reg_[0];
                dst_ptr = reinterpret_cast<uint32_t*>(&smemoutput[idx + 8 * BN / 2]);
                dst_ptr[0] = reg_[1];
            }
        }
        __syncthreads();

#pragma unroll
        for (int subk = 0; subk < WN / 2; ++subk){
            for (int j = 0; j < 4; ++j){
                const uint row =  m_idx + subk + i * WN / 2;
                const uint gemm_i =  n_idx + j*32;
                const int n = fastdiv(gemm_i, param.OHOW_fastdiv);
                const int col = fastmodulo(gemm_i, param.OHOW_fastdiv);
                if(n < param.n && row < param.k && col < param.Oh * param.Ow){
                    const uint outOffset = n * param.k * param.Oh * param.Ow + row * param.Oh * param.Ow + col;
                    uint idx = output_lds_addr + subk + j*32*BN/2;
                    idx = idx ^ ((idx & 0b1110000000) >> 4);
                    output[outOffset] = smemoutput[idx];
                }
            }
        }
    }
#else
    GGML_UNUSED(input);
    GGML_UNUSED(kernel);
    GGML_UNUSED(output);
    GGML_UNUSED(param);
    NO_DEVICE_CODE;
#endif
}


#define NUM_VARIANTS 4

/*
  conv_shapes[][0]: ne_input=[384,512,256,1],ne_kernel=[3,3,256,256]
  conv_shapes[][1]: ne_input=[96,128,512,1],ne_kernel=[3,3,512,512]
  conv_shapes[][2]: ne_input=[192,256,512,1git diff],ne_kernel=[3,3,512,512]
*/
constexpr static int conv_shapes[][NUM_VARIANTS] = {
    { 128, 128,  128, 256 }, // BM
    { 256,  128,  256, 128 }, // BN
    { 8, 8, 8, 8 }, // BK
    { 128, 64,  32, 128   }, // WM
    { 32,  32 ,  256, 32   }, // WN
    { 2,   2,  1, 1   }, // WNITER
    { 8,   4,  4, 4  }, // TM
    { 8,   4,  8, 8   }, // TN
    { 256,  256, 128, 256}	    //  NUM_THREADS
};

template <typename T, unsigned int CONV_SHAPE>
static void conv2d_implicit_cuda(const float * X_D, const T * K_D, float * Y_D, const param_t P, hipStream_t st) {

    const uint BM = conv_shapes[0][CONV_SHAPE];
    const uint BN = conv_shapes[1][CONV_SHAPE];
    const uint BK = conv_shapes[2][CONV_SHAPE];
    const uint WM = conv_shapes[3][CONV_SHAPE];
    const uint WN = conv_shapes[4][CONV_SHAPE];
    const uint WNITER = conv_shapes[5][CONV_SHAPE];
    const uint TM = conv_shapes[6][CONV_SHAPE];
    const uint TN = conv_shapes[7][CONV_SHAPE];
    const uint NUM_THREADS = conv_shapes[8][CONV_SHAPE];
    int blockx = ((P.Oh * P.Ow + BM - 1) / BM); // blockx  number
    int blocky = (P.k + BN-1) / BN;             // blocky  number
    int blockz = P.n;                           // blockz  number
    int thready = 1;   // thready number per block
    int threadz = 1;   // threadz number per block
    dim3 thblock(NUM_THREADS, thready, threadz);
    dim3 grid(blockx, blocky, blockz);

    conv2d_implicit_kernel<T, BM, BN, BK, WM, WN,
          WNITER, TM, TN, NUM_THREADS, 1, false, 0><<<grid, thblock, 0, st>>>(X_D, K_D, Y_D, P);
}

static void conv2d_implicit_cuda_f16(ggml_backend_cuda_context & ctx, const float * X_D, const half * K_D, float * Y_D, int cc, const param_t P, hipStream_t st) {

    if (GGML_CUDA_CC_IS_NVIDIA(cc) && ampere_mma_available(cc) && P.c % 8 == 0 && (P.r > 1 || P.s > 1)) {

        int id = ggml_cuda_get_device();

        int64_t ne = P.c * P.h * P.w * P.n;
        int64_t ne00 = P.c;
        int64_t ne01 = P.h * P.w;
        ggml_cuda_pool_alloc<half> input_f16(ctx.pool(id), ne);

        dim3 dimGrid( (ne01 + CUDA_NCHW_2_NHWC_TILE_DIM - 1) / CUDA_NCHW_2_NHWC_TILE_DIM,
                      (ne00 + CUDA_NCHW_2_NHWC_TILE_DIM - 1) / CUDA_NCHW_2_NHWC_TILE_DIM,
                      (ne/(ne00*ne01) + CUDA_NCHW_2_NHWC_BLOCK_NM - 1) / CUDA_NCHW_2_NHWC_BLOCK_NM) ;
        dim3 dimBlock(CUDA_NCHW_2_NHWC_TILE_DIM,CUDA_NCHW_2_NHWC_BLOCK_ROWS, 1);
        NCHW2NHWC<float, half><<<dimGrid, dimBlock, 0, st>>>(X_D, input_f16.get(), ne, ne00, ne01);

        ne = P.c * P.r * P.s * P.k;
        ne01 = P.r * P.s;
        ggml_cuda_pool_alloc<half> kernel_f16(ctx.pool(id), ne);
        dim3 dimGrid1((ne01 + CUDA_NCHW_2_NHWC_TILE_DIM - 1) / CUDA_NCHW_2_NHWC_TILE_DIM,
                      (ne00 + CUDA_NCHW_2_NHWC_TILE_DIM - 1) / CUDA_NCHW_2_NHWC_TILE_DIM,
                      (ne/(ne00*ne01) + CUDA_NCHW_2_NHWC_BLOCK_NM - 1) / CUDA_NCHW_2_NHWC_BLOCK_NM) ;
        NCHW2NHWC<half, half><<<dimGrid1, dimBlock, 0, st>>>(K_D, kernel_f16.get(), ne, ne00, ne01);

        const half *X_H = input_f16.get();
        const half *K_H = kernel_f16.get();
        ggml_cuda_pool_alloc<half> Y_H(ctx.pool(id), P.k * P.Oh * P.Ow * P.n);

        constexpr unsigned int BM_dim = 256;
        constexpr unsigned int BN_dim = 256;
        constexpr unsigned int BK_dim = 32;

        constexpr unsigned int WARPS_PER_BLOCK_M = 2;
        constexpr unsigned int WARPS_PER_BLOCK_N = 4;
        constexpr unsigned int WARPS_PER_BLOCK_K = 4;

        constexpr unsigned int WM_dim = BM_dim / WARPS_PER_BLOCK_M;
        constexpr unsigned int WN_dim = BN_dim / WARPS_PER_BLOCK_N;
        constexpr unsigned int WK_dim = BK_dim / WARPS_PER_BLOCK_K;
        const unsigned int BlocksM =  (P.n * P.Oh * P.Ow + BM_dim - 1) / BM_dim;
        const unsigned int BlocksN =  (P.k + BN_dim - 1) / BN_dim;
        constexpr unsigned int ThreadsM = WARPS_PER_BLOCK_M;
        constexpr unsigned int ThreadsN = WARPSIZE * WARPS_PER_BLOCK_N;
        constexpr unsigned int NumThreads = ThreadsM * ThreadsN;
        const unsigned int shmem_bytes = (BM_dim * BK_dim + BK_dim * BN_dim) * 2 * sizeof(half);

        hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d_implicit_kernel<BM_dim), BN_dim, BK_dim, WM_dim, WN_dim, WK_dim, NumThreads>,
               hipFuncAttributeMaxDynamicSharedMemorySize,    65536); // set shared memory limit to 64KB which is maximum for sm_75
        dim3 gridDim(BlocksN, BlocksM);
        dim3 blockDim(ThreadsN, ThreadsM);

        conv2d_implicit_kernel<BM_dim, BN_dim, BK_dim,
            WM_dim, WN_dim, WK_dim, NumThreads>
            <<<gridDim, blockDim, shmem_bytes, st>>>(X_H, K_H, Y_H.get(), P);
        const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(GGML_TYPE_F16);
        to_fp32_cuda(Y_H.get(), Y_D, P.k * P.Oh * P.Ow * P.n, st);
    } else{
       conv2d_implicit_cuda<half, 1>(X_D, K_D, Y_D, P, st);
    }

}

static void conv2d_implicit_cuda_f32(ggml_backend_cuda_context & ctx, const float * X_D, const float * K_D, float * Y_D, int cc, const param_t P, hipStream_t st) {
    conv2d_implicit_cuda<float, 1>(X_D, K_D, Y_D, P, st);
    GGML_UNUSED(ctx);
    GGML_UNUSED(cc);
}

void ggml_cuda_op_conv2d_implicit(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * kernel = dst->src[0];
    const ggml_tensor * input  = dst->src[1];
    float *             K_D    = (float *) kernel->data;
    const float *       X_D    = (const float *) input->data;
    float *             Y_D    = (float *) dst->data;

    GGML_ASSERT(ggml_is_contiguous(kernel));
    GGML_ASSERT(kernel->type == GGML_TYPE_F16 || kernel->type == GGML_TYPE_F32);


    hipStream_t st = ctx.stream();
    const int cc            = ggml_cuda_info().devices[ctx.device].cc;

    const int32_t * p    = (const int32_t *) dst->op_params;
    const uint       ST_X = p[0];  // stride_x
    const uint       ST_Y = p[1];  // stride_y
    const uint       PD_X = p[2];  // padding_x
    const uint       PD_Y = p[3];  // padding_y
    const uint       DL_X = p[4];  // dilation_x
    const uint       DL_Y = p[5];  // dilation_y
    // const int       LT   = p[6];  // layout

    // GGML_ASSERT(LT == 0 || LT == 1);

    // same number of input channels
    // GGML_ASSERT(LT == 0 ? input->ne[0] == kernel->ne[0] : input->ne[2] == kernel->ne[2]);
    // No cwhn
    GGML_ASSERT(p[6] == false);

    const uint IW = input->ne[0];   // input_w
    const uint IH = input->ne[1];   // input_h
    const uint OW = dst->ne[0];     // output_w
    const uint OH = dst->ne[1];     // output_h
    const uint KW = kernel->ne[0];  // kernel_w
    const uint KH = kernel->ne[1];  // kernel_h
    const uint IC = input->ne[2];   // input_channels

    const uint OC = kernel->ne[3];  // ouptut_chanles
    const uint B  = input->ne[3];   // n_batches

    param_t params = { B, IC, IH, IW, OC, KH, KW, ST_Y, ST_X, PD_Y, PD_X, DL_Y, DL_X, OH, OW,
                      init_fastdiv_values(KW*IC),
                      init_fastdiv_values(OW),
                      init_fastdiv_values(IC),
                      init_fastdiv_values(KW*KH),
                      init_fastdiv_values(KW),
                      init_fastdiv_values(OW*OH)};

    if (kernel->type == GGML_TYPE_F16) {
        conv2d_implicit_cuda_f16(ctx, X_D, (half *) K_D, Y_D, cc, params, st);
    } else {
        conv2d_implicit_cuda_f32(ctx, X_D, K_D, Y_D, cc, params, st);
    }
}

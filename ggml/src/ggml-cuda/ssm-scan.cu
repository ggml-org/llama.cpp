#include "hip/hip_runtime.h"
#if !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11070
#define USE_CUB
#endif // !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11070

#ifdef USE_CUB
#include <hipcub/hipcub.hpp>
using namespace hipcub;
#endif // USE_CUB

#include "ssm-scan.cuh"

template <size_t splitD, size_t N>
__global__ void __launch_bounds__(splitD, 2)
    ssm_scan_f32(const float *__restrict__ src0, const float *__restrict__ src1, const float *__restrict__ src2,
                 const float *__restrict__ src3, const float *__restrict__ src4, const float *__restrict__ src5,
                 const int src0_nb1, const int src0_nb2, const int src1_nb1, const int src1_nb2,
                 const int src1_nb3, const int src2_nb1, const int src2_nb2, const int src3_nb1,
                 const int src4_nb1, const int src4_nb2, const int src5_nb1, const int src5_nb2,
                 float *__restrict__ dst, const int64_t L)
{

    const float *s0_block = (const float *)((const char *)src0 + blockIdx.x * src0_nb2 + blockIdx.y * splitD * src0_nb1);
    const float *x_block = (const float *)((const char *)src1 + (blockIdx.x * src1_nb2) + blockIdx.y * splitD * sizeof(float));
    const float *dt_block = (const float *)((const char *)src2 + (blockIdx.x * src2_nb2) + blockIdx.y * splitD * sizeof(float));
    const float *A_block = (const float *)((const char *)src3 + blockIdx.y * splitD * src3_nb1);
    const float *B_block = (const float *)((const char *)src4 + (blockIdx.x * src4_nb2));
    const float *C_block = (const float *)((const char *)src5 + (blockIdx.x * src5_nb2));
    float *y_block = (float *)((char *)dst + (blockIdx.x * src1_nb2) + blockIdx.y * splitD * sizeof(float));
    float *s_block = (float *)((char *)dst + src1_nb3 + blockIdx.x * src0_nb2 + blockIdx.y * splitD * src0_nb1);

    const int stride_x = src1_nb1 / sizeof(float);
    const int stride_dt = src2_nb1 / sizeof(float);
    const int stride_B = src4_nb1 / sizeof(float);
    const int stride_C = src5_nb1 / sizeof(float);
    const int stride_y = stride_x;

    float regA[N];
    float regs0[N];

    __shared__ float smemB[N];
    __shared__ float smemC[N];

#ifdef USE_CUB
    using BlockLoadA = hipcub::BlockLoad<float, splitD, N, hipcub::BLOCK_LOAD_VECTORIZE>;
    using BlockLoadS0 = hipcub::BlockLoad<float, splitD, N, hipcub::BLOCK_LOAD_VECTORIZE>;
    using BlockStoreS = hipcub::BlockStore<float, splitD, N, hipcub::BLOCK_STORE_VECTORIZE>;

    __shared__ typename BlockLoadA::TempStorage block_load_tempA;
    __shared__ typename BlockLoadS0::TempStorage block_load_tempS0;
    __shared__ typename BlockStoreS::TempStorage block_store_tempS;

    BlockLoadA(block_load_tempA).Load(A_block, regA);
    BlockLoadS0(block_load_tempS0).Load(s0_block, regs0);
#else
    const int stride_s0 = src0_nb1 / sizeof(float);
    const int stride_A = src3_nb1 / sizeof(float);
#pragma unroll
    for (int j = 0; j < N; ++j)
    {
        regA[j] = A_block[threadIdx.x * stride_A + j];
        regs0[j] = s0_block[threadIdx.x * stride_s0 + j];
    }
#endif

    for (int i = 0; i < L; i++)
    {
        if (threadIdx.x < N)
        {
            smemB[threadIdx.x] = B_block[i * stride_B + threadIdx.x];
            smemC[threadIdx.x] = C_block[i * stride_C + threadIdx.x];
        }
        __syncthreads();

        float dt_soft_plus = dt_block[i * stride_dt + threadIdx.x];
        if (dt_soft_plus <= 20.0f)
        {
            dt_soft_plus = log1pf(expf(dt_soft_plus));
        }
        float x_dt = x_block[i * stride_x + threadIdx.x] * dt_soft_plus;

        float sumf = 0.0f;
#pragma unroll
        for (int j = 0; j < N; j++)
        {
            float state = regs0[j] * expf(dt_soft_plus * regA[j]) + smemB[j] * x_dt;
            sumf += state * smemC[j];
            regs0[j] = state;
        }
        y_block[i * stride_y + threadIdx.x] = sumf;
    }

#ifdef USE_CUB
    BlockStoreS(block_store_tempS).Store(s_block, regs0);
#else
    const int stride_s = stride_s0;
#pragma unroll
    for (int j = 0; j < N; ++j)
    {
        s_block[threadIdx.x * stride_s + j] = regs0[j];
    }
#endif
}

template <size_t splitD, size_t N>
__global__ void __launch_bounds__(splitD, 2)
    ssm_scan_single_step_f32(const float *__restrict__ src0, const float *__restrict__ src1, const float *__restrict__ src2,
                             const float *__restrict__ src3, const float *__restrict__ src4, const float *__restrict__ src5,
                             const int src0_nb1, const int src0_nb2, const int src1_nb2,
                             const int src1_nb3, const int src2_nb2, const int src3_nb1,
                             const int src4_nb2, const int src5_nb2,
                             float *__restrict__ dst)
{
    const float *s0_block = (const float *)((const char *)src0 + blockIdx.x * src0_nb2 + blockIdx.y * splitD * src0_nb1);
    const float *x_block = (const float *)((const char *)src1 + (blockIdx.x * src1_nb2) + blockIdx.y * splitD * sizeof(float));
    const float *dt_block = (const float *)((const char *)src2 + (blockIdx.x * src2_nb2) + blockIdx.y * splitD * sizeof(float));
    const float *A_block = (const float *)((const char *)src3 + blockIdx.y * splitD * src3_nb1);
    const float *B_block = (const float *)((const char *)src4 + (blockIdx.x * src4_nb2));
    const float *C_block = (const float *)((const char *)src5 + (blockIdx.x * src5_nb2));
    float *y_block = (float *)((char *)dst + (blockIdx.x * src1_nb2) + blockIdx.y * splitD * sizeof(float));
    float *s_block = (float *)((char *)dst + src1_nb3 + blockIdx.x * src0_nb2 + blockIdx.y * splitD * src0_nb1);

    float regA[N];
    float regs0[N];

    __shared__ float smemB[N];
    __shared__ float smemC[N];

#ifdef USE_CUB
    using BlockLoadA = hipcub::BlockLoad<float, splitD, N, hipcub::BLOCK_LOAD_VECTORIZE>;
    using BlockLoadS0 = hipcub::BlockLoad<float, splitD, N, hipcub::BLOCK_LOAD_VECTORIZE>;
    using BlockStoreS = hipcub::BlockStore<float, splitD, N, hipcub::BLOCK_STORE_VECTORIZE>;

    __shared__ typename BlockLoadA::TempStorage block_load_tempA;
    __shared__ typename BlockLoadS0::TempStorage block_load_tempS0;
    __shared__ typename BlockStoreS::TempStorage block_store_tempS;

    BlockLoadA(block_load_tempA).Load(A_block, regA);
    BlockLoadS0(block_load_tempS0).Load(s0_block, regs0);
#else
    const int stride_s0 = src0_nb1 / sizeof(float);
    const int stride_A = src3_nb1 / sizeof(float);
#pragma unroll
    for (int j = 0; j < N; ++j)
    {
        regA[j] = A_block[threadIdx.x * stride_A + j];
        regs0[j] = s0_block[threadIdx.x * stride_s0 + j];
    }
#endif

    if (threadIdx.x < N)
    {
        smemB[threadIdx.x] = B_block[threadIdx.x];
        smemC[threadIdx.x] = C_block[threadIdx.x];
    }
    __syncthreads();

    {
        float dt_soft_plus = dt_block[threadIdx.x];
        if (dt_soft_plus <= 20.0f)
        {
            dt_soft_plus = log1pf(expf(dt_soft_plus));
        }
        float x_dt = x_block[threadIdx.x] * dt_soft_plus;
        float sumf = 0.0f;
#pragma unroll
        for (int j = 0; j < N; j++)
        {
            float state = regs0[j] * expf(dt_soft_plus * regA[j]) + smemB[j] * x_dt;
            sumf += state * smemC[j];
            regs0[j] = state;
        }
        y_block[threadIdx.x] = sumf;
    }

#ifdef USE_CUB
    BlockStoreS(block_store_tempS).Store(s_block, regs0);
#else
    const int stride_s = s0;
#pragma unroll
    for (int j = 0; j < N; ++j)
    {
        s_block[threadIdx.x * stride_s + j] = regs0[j];
    }
#endif
}

static void ssm_scan_f32_cuda(const float *src0, const float *src1, const float *src2, const float *src3,
                              const float *src4, const float *src5, const int src0_nb1, const int src0_nb2,
                              const int src1_nb1, const int src1_nb2, const int src1_nb3,
                              const int src2_nb1, const int src2_nb2, const int src3_nb1,
                              const int src4_nb1, const int src4_nb2, const int src5_nb1, const int src5_nb2,
                              float *dst, const int64_t N, const int64_t D, const int64_t L, const int64_t B,
                              hipStream_t stream)
{
    const int threads = 128;
    // todo: consider D cannot be divided,does this situation exist?
    GGML_ASSERT(D % threads == 0);
    const dim3 blocks(B, (D + threads - 1) / threads, 1);
    if (N == 16)
    {
        if (L > 1)
        {
            ssm_scan_f32<threads, 16><<<blocks, threads, 0, stream>>>(
                src0, src1, src2, src3, src4, src5, src0_nb1, src0_nb2, src1_nb1, src1_nb2, src1_nb3,
                src2_nb1, src2_nb2, src3_nb1, src4_nb1, src4_nb2, src5_nb1, src5_nb2, dst, L);
        }
        else
        {
            ssm_scan_single_step_f32<threads, 16><<<blocks, threads, 0, stream>>>(
                src0, src1, src2, src3, src4, src5, src0_nb1, src0_nb2, src1_nb2,
                src1_nb3, src2_nb2, src3_nb1,
                src4_nb2, src5_nb2,
                dst);
        }
    }
    else
    {
        GGML_ABORT("doesn't support N!=16.");
    }
}

void ggml_cuda_op_ssm_scan(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0];  // s
    const struct ggml_tensor * src1 = dst->src[1];  // x
    const struct ggml_tensor * src2 = dst->src[2];  // dt
    const struct ggml_tensor * src3 = dst->src[3];  // A
    const struct ggml_tensor * src4 = dst->src[4];  // B
    const struct ggml_tensor * src5 = dst->src[5];  // C

    //   const int64_t d_state = src0->ne[0];
    //   const int64_t d_inner = src0->ne[1];
    //   const int64_t l = src1->ne[1];
    //   const int64_t b = src0->ne[2];

    const int64_t nc  = src0->ne[0];  // d_state
    const int64_t nr  = src0->ne[1];  // d_inner
    const int64_t n_t = src1->ne[1];  // number of tokens per sequence
    const int64_t n_s = src0->ne[2];  // number of sequences in the batch

    GGML_ASSERT(ggml_nelements(src1) + ggml_nelements(src0) == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(float));
    GGML_ASSERT(src4->nb[0] == sizeof(float));
    GGML_ASSERT(src5->nb[0] == sizeof(float));
    // required for the dot product between s and C
    GGML_ASSERT(src0->nb[1] == src0->ne[0] * sizeof(float));
    // required for per-sequence offsets for states
    GGML_ASSERT(src0->nb[2] == src0->ne[0] * src0->ne[1] * sizeof(float));
    // required to get correct offset for state destination (i.e. src1->nb[3])
    GGML_ASSERT(src1->nb[3] == src1->ne[0] * src1->ne[1] * src1->ne[2] * sizeof(float));

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    const float * src2_d = (const float *) src2->data;
    const float * src3_d = (const float *) src3->data;
    const float * src4_d = (const float *) src4->data;
    const float * src5_d = (const float *) src5->data;
    float *       dst_d  = (float *) dst->data;
    hipStream_t  stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    ssm_scan_f32_cuda(src0_d, src1_d, src2_d, src3_d, src4_d, src5_d, src0->nb[1], src0->nb[2],
                      src1->nb[1], src1->nb[2], src1->nb[3], src2->nb[1], src2->nb[2], src3->nb[1],
                      src4->nb[1], src4->nb[2], src5->nb[1], src5->nb[2], dst_d, nc, nr, n_t, n_s, stream);
}
